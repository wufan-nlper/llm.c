#include "hip/hip_runtime.h"
/*
Kernels for layernorm backward pass.

Compile example:
nvcc -O3 --use_fast_math layernorm_backward.cu -o layernorm_backward

version 1 is naive port from CPU code to kernel: parallelizes over B,T, loops over C
./layernorm_backward 1
*/

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <assert.h>
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include "common.h"

// ----------------------------------------------------------------------------
// CPU code reference

void layernorm_forward_cpu(float *out, float *mean, float *rstd,
                           float *inp, float *weight, float *bias,
                           int B, int T, int C) {
    // reference: https://pytorch.org/docs/stable/generated/torch.nn.LayerNorm.html
    // both inp and out are (B,T,C) of the activations
    // mean and rstd are (B,T) buffers, to be used later in backward pass
    // at each position (b,t) of the input, the C-dimensional vector
    // of activations gets normalized, then scaled and shifted
    float eps = 1e-5f;
    for (int b = 0; b < B; b++) {
        for (int t = 0; t < T; t++) {
            // seek to the input position inp[b,t,:]
            float *x = inp + b * T * C + t * C;
            // calculate the mean
            float m = 0.0f;
            for (int i = 0; i < C; i++) {
                m += x[i];
            }
            m = m / C;
            // calculate the variance (without any bias correction)
            float v = 0.0f;
            for (int i = 0; i < C; i++) {
                float xshift = x[i] - m;
                v += xshift * xshift;
            }
            v = v / C;
            // calculate the rstd (reciprocal standard deviation)
            float s = 1.0f / sqrtf(v + eps);
            // seek to the output position in out[b,t,:]
            float *out_bt = out + b * T * C + t * C;
            for (int i = 0; i < C; i++) {
                float n = (s * (x[i] - m)); // normalize
                float o = n * weight[i] + bias[i]; // scale and shift
                out_bt[i] = o; // write
            }
            // cache the mean and rstd for the backward pass later
            mean[b * T + t] = m;
            rstd[b * T + t] = s;
        }
    }
}

void layernorm_backward_cpu(float *dinp, float *dweight, float *dbias,
                            float *dout, float *inp, float *weight, float *mean, float *rstd,
                            int B, int T, int C) {
    for (int b = 0; b < B; b++) {
        for (int t = 0; t < T; t++) {
            float *dout_bt = dout + b * T * C + t * C;
            float *inp_bt = inp + b * T * C + t * C;
            float *dinp_bt = dinp + b * T * C + t * C;
            float mean_bt = mean[b * T + t];
            float rstd_bt = rstd[b * T + t];

            // first: two reduce operations
            float dnorm_mean = 0.0f;
            float dnorm_norm_mean = 0.0f;
            for (int i = 0; i < C; i++) {
                float norm_bti = (inp_bt[i] - mean_bt) * rstd_bt;
                float dnorm_i = weight[i] * dout_bt[i];
                dnorm_mean += dnorm_i;
                dnorm_norm_mean += dnorm_i * norm_bti;
            }
            dnorm_mean = dnorm_mean / C;
            dnorm_norm_mean = dnorm_norm_mean / C;

            // now iterate again and accumulate all the gradients
            for (int i = 0; i < C; i++) {
                float norm_bti = (inp_bt[i] - mean_bt) * rstd_bt;
                float dnorm_i = weight[i] * dout_bt[i];
                // gradient contribution to bias
                dbias[i] += dout_bt[i];
                // gradient contribution to weight
                dweight[i] += norm_bti * dout_bt[i];
                // gradient contribution to input
                float dval = 0.0f;
                dval += dnorm_i; // term 1
                dval -= dnorm_mean; // term 2
                dval -= norm_bti * dnorm_norm_mean; // term 3
                dval *= rstd_bt; // final scale
                dinp_bt[i] += dval;
            }
        }
    }
}

// ----------------------------------------------------------------------------
// GPU kernels

// super naive kernel that just parallelizes over B,T and loops over C
__global__ void layernorm_backward_kernel1(float *dinp, float *dweight, float *dbias,
                                           float *dout, float *inp, float *weight, float *mean, float *rstd,
                                           int B, int T, int C) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= B * T) return;
    int b = idx / T;
    int t = idx % T;

    float *dout_bt = dout + b * T * C + t * C;
    float *inp_bt = inp + b * T * C + t * C;
    float *dinp_bt = dinp + b * T * C + t * C;
    float mean_bt = mean[b * T + t];
    float rstd_bt = rstd[b * T + t];

    // first: two reduce operations
    float dnorm_mean = 0.0f;
    float dnorm_norm_mean = 0.0f;
    for (int i = 0; i < C; i++) {
        float norm_bti = (inp_bt[i] - mean_bt) * rstd_bt;
        float dnorm_i = weight[i] * dout_bt[i];
        dnorm_mean += dnorm_i;
        dnorm_norm_mean += dnorm_i * norm_bti;
    }
    dnorm_mean = dnorm_mean / C;
    dnorm_norm_mean = dnorm_norm_mean / C;

    // now iterate again and accumulate all the gradients
    for (int i = 0; i < C; i++) {
        float norm_bti = (inp_bt[i] - mean_bt) * rstd_bt;
        float dnorm_i = weight[i] * dout_bt[i];
        // gradient contribution to bias
        atomicAdd(&dbias[i], dout_bt[i]);
        // gradient contribution to weight
        atomicAdd(&dweight[i], norm_bti * dout_bt[i]);
        // gradient contribution to input
        float dval = 0.0f;
        dval += dnorm_i; // term 1
        dval -= dnorm_mean; // term 2
        dval -= norm_bti * dnorm_norm_mean; // term 3
        dval *= rstd_bt; // final scale
        dinp_bt[i] += dval;
    }
}

// ----------------------------------------------------------------------------
// kernel launchers

void layernorm_backward1(float *dinp, float *dweight, float *dbias,
                         float *dout, float *inp, float *weight, float *mean, float *rstd,
                         int B, int T, int C, const int block_size) {
    const int N = B * T;
    const int grid_size = ceil_div(N, block_size);
    layernorm_backward_kernel1<<<grid_size, block_size>>>(dinp, dweight, dbias, dout, inp, weight, mean, rstd, B, T, C);
}

// kernel version dispatch
void layernorm_backward(int kernel_num,
                        float *dinp, float *dweight, float *dbias,
                        float *dout, float *inp, float *weight, float *mean, float *rstd,
                        int B, int T, int C,
                        const int block_size) {
    switch (kernel_num) {
        case 1:
            layernorm_backward1(dinp, dweight, dbias, dout, inp, weight, mean, rstd, B, T, C, block_size);
            break;
        default:
            printf("Invalid kernel number\n");
            exit(1);
    }
}

// ----------------------------------------------------------------------------

int main(int argc, char **argv) {
    srand(0);

    int B = 8;
    int T = 1024;
    int C = 768;

    int deviceIdx = 0;
    cudaCheck(hipSetDevice(deviceIdx));

    // first do the forward pass in CPU
    float *out = (float *) malloc(B * T * C * sizeof(float));
    float *mean = (float *) malloc(B * T * sizeof(float));
    float *rstd = (float *) malloc(B * T * sizeof(float));
    float *inp = make_random_float(B * T * C);
    float *weight = make_random_float(C);
    float *bias = make_random_float(C);
    layernorm_forward_cpu(out, mean, rstd, inp, weight, bias, B, T, C);

    // now do the backward pass, again on CPU
    float *dout = make_random_float(B * T * C);
    float *dinp = make_zeros_float(B * T * C);
    float *dweight = make_zeros_float(C);
    float *dbias = make_zeros_float(C);
    layernorm_backward_cpu(dinp, dweight, dbias, dout, inp, weight, mean, rstd, B, T, C);

    // the above calculations act as the reference
    // now let's do the same on the GPU

    // read kernel_num from command line
    int kernel_num = 2;
    if (argc > 1) {
        kernel_num = atoi(argv[1]);
    }
    printf("Using kernel %d\n", kernel_num);

    // move all the variables we need for backward pass onto the GPU
    float *d_dinp;
    float *d_dweight;
    float *d_dbias;
    float *d_dout;
    float *d_inp;
    float *d_weight;
    float *d_mean;
    float *d_rstd;
    cudaCheck(hipMalloc(&d_dinp, B * T * C * sizeof(float)));
    cudaCheck(hipMalloc(&d_dweight, C * sizeof(float)));
    cudaCheck(hipMalloc(&d_dbias, C * sizeof(float)));
    cudaCheck(hipMalloc(&d_dout, B * T * C * sizeof(float)));
    cudaCheck(hipMalloc(&d_inp, B * T * C * sizeof(float)));
    cudaCheck(hipMalloc(&d_weight, C * sizeof(float)));
    cudaCheck(hipMalloc(&d_mean, B * T * sizeof(float)));
    cudaCheck(hipMalloc(&d_rstd, B * T * sizeof(float)));
    // copy over the "inputs" to the backward call
    cudaCheck(hipMemcpy(d_dout, dout, B * T * C * sizeof(float), hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(d_inp, inp, B * T * C * sizeof(float), hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(d_weight, weight, C * sizeof(float), hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(d_mean, mean, B * T * sizeof(float), hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(d_rstd, rstd, B * T * sizeof(float), hipMemcpyHostToDevice));
    // init the "outputs" of the backward call to zeros
    cudaCheck(hipMemset(d_dinp, 0, B * T * C * sizeof(float)));
    cudaCheck(hipMemset(d_dweight, 0, C * sizeof(float)));
    cudaCheck(hipMemset(d_dbias, 0, C * sizeof(float)));

    // launch the kernel
    const int block_size = 256;
    layernorm_backward(kernel_num, d_dinp, d_dweight, d_dbias, d_dout, d_inp, d_weight, d_mean, d_rstd, B, T, C,
                       block_size);

    // check the correctness of the kernel
    printf("Checking correctness...\n");
    printf("dinp:\n");
    validate_result(d_dinp, dinp, "dinp", B * T * C, 1e-3f);
    printf("dweight:\n");
    validate_result(d_dweight, dweight, "dweight", C, 1e-3f);
    printf("dbias:\n");
    validate_result(d_dbias, dbias, "dbias", C, 1e-3f);

    // now time the kernel
    int block_sizes[] = {32, 64, 128, 256, 512, 1024};
    for (int j = 0; j < sizeof(block_sizes) / sizeof(int); j++) {
        int block_size = block_sizes[j];
        int repeat_times = 100;
        float elapsed_time = benchmark_kernel(repeat_times, layernorm_backward, kernel_num,
                                              d_dinp, d_dweight, d_dbias, d_dout, d_inp, d_weight, d_mean, d_rstd,
                                              B, T, C, block_size);
        printf("block_size %4d time %.4f ms\n", block_size, elapsed_time);
    }

    // cleanups
    free(out);
    free(mean);
    free(rstd);
    free(inp);
    free(weight);
    free(bias);
    free(dout);
    free(dinp);
    free(dweight);
    free(dbias);
    cudaCheck(hipFree(d_dinp));
    cudaCheck(hipFree(d_dweight));
    cudaCheck(hipFree(d_dbias));
    cudaCheck(hipFree(d_dout));
    cudaCheck(hipFree(d_inp));
    cudaCheck(hipFree(d_weight));
    cudaCheck(hipFree(d_mean));
    cudaCheck(hipFree(d_rstd));

    return 0;
}