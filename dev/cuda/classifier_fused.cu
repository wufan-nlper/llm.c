#include "hip/hip_runtime.h"
/*  Kernels for fused forward/backward classifier part
This fuses softmax, crossentropy, and logit gradients into a single pass, so we don't have to write unnecessary
(B, T, V) tensors. Such an operation is only possible if `dloss` can be known beforehand, which doesn't seem like
much of a restriction: In pretraining, it is just a constant 1/batch_size tensor, for fine-tuning we might zero
out the input prompt, but that is known in advance.

Compile example:
nvcc -O3 --use_fast_math classifier_fused.cu -o classifier_fused
*/

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include "common.h"

// ----------------------------------------------------------------------------
// CPU code reference

void softmax_forward_cpu(float *out, const float *inp, int N, int C) {
    // inp is (N, C)
    // out is (N, C), each row of inp will get softmaxed
    for (int i = 0; i < N; i++) {
        const float *inp_row = inp + i * C;
        float *out_row = out + i * C;

        float maxval = -INFINITY;
        for (int j = 0; j < C; j++) {
            if (inp_row[j] > maxval) {
                maxval = inp_row[j];
            }
        }
        double sum = 0.0;
        for (int j = 0; j < C; j++) {
            out_row[j] = expf(inp_row[j] - maxval);
            sum += out_row[j];
        }
        for (int j = 0; j < C; j++) {
            out_row[j] /= sum;
        }
    }
}


void crossentropy_forward_cpu(float *losses,
                              const float *probs, const int *targets,
                              int B, int T, int V) {
    // output: losses is (B,T) of the individual losses at each position
    // input: probs are (B,T,V) of the probabilities
    // input: targets is (B,T) of integers giving the correct index in logits
    for (int b = 0; b < B; b++) {
        for (int t = 0; t < T; t++) {
            // loss = -log(probs[target])
            const float *probs_bt = probs + b * T * V + t * V;
            int ix = targets[b * T + t];
            losses[b * T + t] = -logf(probs_bt[ix]);
        }
    }
}

void crossentropy_softmax_backward_cpu(float *dlogits,
                                       const float *dlosses, const float *probs, const int *targets,
                                       int B, int T, int V) {
    // backwards through both softmax and crossentropy
    for (int b = 0; b < B; b++) {
        for (int t = 0; t < T; t++) {
            float *dlogits_bt = dlogits + b * T * V + t * V;
            const float *probs_bt = probs + b * T * V + t * V;
            float dloss = dlosses[b * T + t];
            int ix = targets[b * T + t];
            for (int i = 0; i < V; i++) {
                float p = probs_bt[i];
                float indicator = i == ix ? 1.0f : 0.0f;
                dlogits_bt[i] = (p - indicator) * dloss;
            }
        }
    }
}

// ----------------------------------------------------------------------------
// GPU kernels

struct SoftmaxParams {
    float Scale;
    float Offset;
};
namespace cg = cooperative_groups;

__device__ SoftmaxParams prepare_softmax(cg::thread_block_tile<32> &warp,
                                         int idx, const float *inp, int V, int P) {
    // one row of inp, i.e. inp[idx, :] of shape (V,)
    const float *x = inp + idx * P;

    float maxval = -INFINITY;
    float sumval = 0.0f;

    for (int i = warp.thread_rank(); i < V; i += warp.size()) {
        float v = x[i];
        float old_maxval = maxval;
        maxval = fmaxf(maxval, v);
        sumval *= expf((old_maxval - maxval));
        sumval += expf(v - maxval);
    }

    float global_maxval = cg::reduce(warp, maxval, cg::greater < float > {});
    sumval *= expf((maxval - global_maxval));

    float sum = cg::reduce(warp, sumval, cg::plus < float > {});
    float norm = 1.f / sum;

    return SoftmaxParams{norm, global_maxval};
}


__global__ void fused_classifier_kernel(float *dlogits, float *losses,
                                        const float *logits, const float *dlosses, const int *targets,
                                        int B, int T, int V, int P) {
    namespace cg = cooperative_groups;
    cg::thread_block block = cg::this_thread_block();
    cg::thread_block_tile<32> warp = cg::tiled_partition<32>(block);
    int idx = blockIdx.x * warp.meta_group_size() + warp.meta_group_rank();
    if (idx >= B * T) {
        return;
    }

    // local indices
    int b = idx / T;
    int t = idx % T;

    auto sp = prepare_softmax(warp, idx, logits, V, P);

    // calculate the probability needed for the loss and update.
    // single-threaded
    if (warp.thread_rank() == 0) {
        int ix = targets[b * T + t];
        float prob = expf(logits[idx * P + ix] - sp.Offset) * sp.Scale;
        losses[b * T + t] = -logf(prob);
    }

    // calculate all the gradients
    for (int i = warp.thread_rank(); i < V; i += warp.size()) {
        float prob = expf(logits[idx * P + i] - sp.Offset) * sp.Scale;
        float *dlogits_bt = dlogits + b * T * P + t * P;
        float dloss = dlosses[b * T + t];
        int ix = targets[b * T + t];
        float p = prob;
        float indicator = i == ix ? 1.0f : 0.0f;
        dlogits_bt[i] = (p - indicator) * dloss;
    }

}

// ----------------------------------------------------------------------------
// kernel launcher

void fused_classifier1(float *dlogits, float *losses,
                       const float *logits, const float *dlosses, const int *targets,
                       int B, int T, int V, int P, int block_size) {
    const int N = B * T;
    const int grid_size = N;
    fused_classifier_kernel<<<grid_size, block_size>>>(dlogits, losses, logits, dlosses, targets, B, T, V, P);
    cudaCheck(hipGetLastError());
}

void fused_classifier(int kernel_num, float *dlogits, float *losses,
                      const float *logits, const float *dlosses, const int *targets,
                      int B, int T, int V, int P, int block_size) {
    switch (kernel_num) {
        case 1:
            fused_classifier1(dlogits, losses, logits, dlosses, targets, B, T, V, P, block_size);
            break;
        default:
            printf("Invalid kernel number\n");
            exit(1);
    }
}

// ----------------------------------------------------------------------------

int main(int argc, char **argv) {
    srand(0);

    int B = 8;
    int T = 1024;
    int V = 50257;
    // padded size
    int P = (V + 63) & ~63;

    int deviceIdx = 0;
    cudaCheck(hipSetDevice(deviceIdx));

    // create host memory of random numbers
    float *logits = make_random_float_01(B * T * V);
    float *probs = (float *) malloc(B * T * V * sizeof(float));
    float *dlogits = (float *) malloc(B * T * V * sizeof(float));
    float *losses = (float *) malloc(B * T * sizeof(float));
    const float *dlosses = make_random_float(B * T);
    const int *targets = make_random_int(B * T, V);

    // make the input less uniformly random: Otherwise, all probabilities will be basically zero,
    // and the tests are not actually meaningful.
    const int *outliers = make_random_int(B * T * 3, V);
    for (int k = 0; k < 3; ++k) {
        for (int j = 0; j < B * T; ++j) {
            logits[j * V + outliers[j * 3 + k]] *= 20;
        }
    }

    // move to GPU
    float *d_logits;
    float *d_dlogits;
    float *d_dlogits_no_pad;
    float *d_losses;
    float *d_dlosses;
    int *d_targets;

    cudaCheck(hipMalloc(&d_dlogits, B * T * P * sizeof(float)));
    cudaCheck(hipMalloc(&d_logits, B * T * P * sizeof(float)));
    cudaCheck(hipMalloc(&d_dlogits_no_pad, B * T * V * sizeof(float)));
    cudaCheck(hipMalloc(&d_targets, B * T * sizeof(int)));
    cudaCheck(hipMalloc(&d_losses, B * T * sizeof(float)));
    cudaCheck(hipMalloc(&d_dlosses, B * T * sizeof(float)));

    // move to GPU
    cudaCheck(hipMemset(d_logits, 0xff, B * T * P * sizeof(float)));
    cudaCheck(hipMemcpy2D(d_logits, P * sizeof(float), logits, V * sizeof(float), V * sizeof(float), B * T,
                           hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(d_dlosses, dlosses, B * T * sizeof(float), hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(d_targets, targets, B * T * sizeof(int), hipMemcpyHostToDevice));

    // read kernel_num from command line
    int kernel_num = 1;
    if (argc > 1) {
        kernel_num = atoi(argv[1]);
    }
    printf("Using kernel %d\n", kernel_num);

    // first check the correctness of the kernel
    softmax_forward_cpu(probs, logits, B * T, V);
    crossentropy_forward_cpu(losses, probs, targets, B, T, V);
    crossentropy_softmax_backward_cpu(dlogits, dlosses, probs, targets, B, T, V);

    // time the kernel at different block sizes
    int block_sizes[] = {32, 64, 128, 256, 512, 1024};

    for (int j = 0; j < sizeof(block_sizes) / sizeof(int); j++) {
        int block_size = block_sizes[j];
        printf("Checking block size %d.\n", block_size);
        fused_classifier(kernel_num, d_dlogits, d_losses, d_logits, d_dlosses, d_targets, B, T, V, P, block_size);
        validate_result(d_losses, losses, "losses", B * T, 1e-4f);
        // undo the padding before we can check for correctness
        cudaCheck(hipMemcpy2D(d_dlogits_no_pad, V * sizeof(float), d_dlogits, P * sizeof(float), V * sizeof(float),
                               B * T, hipMemcpyDeviceToDevice));
        validate_result(d_dlogits_no_pad, dlogits, "dlogits", B * T * V, 1e-4f);
    }

    printf("All results match. Starting benchmarks.\n\n");

    for (int j = 0; j < sizeof(block_sizes) / sizeof(int); j++) {
        int block_size = block_sizes[j];

        int repeat_times = 1000;
        float elapsed_time = benchmark_kernel(repeat_times, fused_classifier,
                                              kernel_num, d_dlogits, d_losses, d_logits, d_dlosses, d_targets,
                                              B, T, V, P, block_size);

        printf("block_size %4d | time %f ms\n", block_size, elapsed_time);
    }

    // free memory
    free((void *) logits);
    free(probs);
    free(dlogits);
    free(losses);
    free((void *) dlosses);
    free((void *) targets);

    cudaCheck(hipFree(d_dlogits));
    cudaCheck(hipFree(d_losses));
    cudaCheck(hipFree(d_logits));
    cudaCheck(hipFree(d_dlosses));
    cudaCheck(hipFree(d_targets));

    return 0;
}