#include "hip/hip_runtime.h"
/*
Kernels for layernorm forward pass.

Compile example:
nvcc -O3 --use_fast_math layernorm_forward.cu -o layernorm_forward

version 1 is naive port from CPU code to kernel: parallelizes over B,T, loops over C
./layernorm_forward 1

version 2 parallelizes over all of B,T,C
./layernorm_forward 2

version 3 uses cooperative groups to parallelize over all of B,T,C
./layernorm_forward 3
*/

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <assert.h>
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include "common.h"

// ----------------------------------------------------------------------------
// CPU code reference

// GPT-2 layernorm forward pass
void layernorm_forward_cpu(float *out, float *mean, float *rstd,
                           const float *inp, const float *weight, const float *bias,
                           int B, int T, int C) {
    float eps = 1e-5f;
    for (int b = 0; b < B; b++) {
        for (int t = 0; t < T; t++) {
            // seek to the input position inp[b,t,:]
            const float *x = inp + b * T * C + t * C;
            // calculate the mean
            float m = 0.0f;
            for (int i = 0; i < C; i++) {
                m += x[i];
            }
            m = m / C;
            // calculate the variance (without any bias correction)
            float v = 0.0f;
            for (int i = 0; i < C; i++) {
                float xshift = x[i] - m;
                v += xshift * xshift;
            }
            v = v / C;
            // calculate the rstd
            float s = 1.0f / sqrtf(v + eps);
            // seek to the output position in out[b,t,:]
            float *out_bt = out + b * T * C + t * C;
            for (int i = 0; i < C; i++) {
                float n = (s * (x[i] - m)); // normalized output
                float o = n * weight[i] + bias[i]; // scale and shift it
                out_bt[i] = o; // write
            }
            // cache the mean and rstd for the backward pass later
            mean[b * T + t] = m;
            rstd[b * T + t] = s;
        }
    }
}

// ----------------------------------------------------------------------------
// GPU kernels

// naive drag and drop implementation into kernel, parallelize over B,T, loop over C
__global__ void layernorm_forward_kernel1(float *out, float *mean, float *rstd,
                                          const float *inp, const float *weight, const float *bias,
                                          int N, int C) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    float eps = 1e-5f;

    if (idx < N) {
        // seek to the input position inp[idx,:]
        const float *x = inp + idx * C;
        // calculate the mean
        float m = 0.0f;
        for (int i = 0; i < C; i++) {
            m += x[i];
        }
        m = m / C;
        // calculate the variance (without any bias correction)
        float v = 0.0f;
        for (int i = 0; i < C; i++) {
            float xshift = x[i] - m;
            v += xshift * xshift;
        }
        v = v / C;
        // calculate the rstd
        float s = 1.0f / sqrtf(v + eps);
        // seek to the output position in out[idx,:]
        float *out_idx = out + idx * C;
        for (int i = 0; i < C; i++) {
            float n = (s * (x[i] - m)); // normalized output
            float o = n * weight[i] + bias[i]; // scale and shift it
            out_idx[i] = o; // write
        }
        // cache the mean and rstd for the backward pass later
        mean[idx] = m;
        rstd[idx] = s;
    }
}

__global__ void mean_kernel(float *mean, const float *inp, int N, int C, int block_size) {
    extern __shared__ float shared[];
    int idx = blockIdx.x; // range [0, B*T)
    int tid = threadIdx.x; // range [0, block_size)
    const float *x = inp + idx * C;
    // thread coarsening
    float sum = 0.0f;
    for (int i = tid; i < C; i += block_size) {
        sum += x[i];
    }
    shared[tid] = sum;
    __syncthreads();
    // reductions
    for (int stride = block_size / 2; stride >= 1; stride /= 2) {
        __syncthreads();
        if (tid < stride) {
            shared[tid] += shared[tid + stride];
        }
    }
    // write the final result (at thread 0) to global memory
    if (tid == 0) {
        mean[idx] = shared[0] / C;
    }
}

__global__ void rstd_kernel(float *rstd, const float *inp, const float *mean, int N, int C, int block_size) {
    extern __shared__ float shared[];
    int idx = blockIdx.x; // range [0, B*T)
    int tid = threadIdx.x; // range [0, block_size)
    const float *x = inp + idx * C;
    float m = mean[idx];
    // thread coarsening
    float sum = 0.0f;
    for (int i = tid; i < C; i += block_size) {
        float diff = x[i] - m;
        sum += diff * diff;
    }
    shared[tid] = sum;
    __syncthreads();
    // reductions
    for (int stride = block_size / 2; stride >= 1; stride /= 2) {
        __syncthreads();
        if (tid < stride) {
            shared[tid] += shared[tid + stride];
        }
    }
    // write the final result (at thread 0) to global memory
    if (tid == 0) {
        rstd[idx] = 1.0f / sqrtf(shared[0] / C + 1e-5f);
    }
}

__global__ void normalization_kernel(float *out, const float *inp, float *mean, float *rstd,
                                     const float *weight, const float *bias, int B, int T, int C) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    int bt = idx / C;
    int c = idx % C;

    float m = mean[bt];
    float s = rstd[bt];
    float xi = inp[idx];
    float n = s * (xi - m);
    float o = n * weight[c] + bias[c];

    out[idx] = o;
}

// ----------------------------------------------------------------------------

__global__ void layernorm_forward_kernel3(float *__restrict__ out, float *__restrict__ mean, float *__restrict__ rstd,
                                          const float *__restrict__ inp, const float *__restrict__ weight,
                                          const float *__restrict__ bias, int N, int C) {
    namespace cg = cooperative_groups;
    cg::thread_block block = cg::this_thread_block();
    cg::thread_block_tile<32> warp = cg::tiled_partition<32>(block);
    int idx = blockIdx.x * warp.meta_group_size() + warp.meta_group_rank();
    if (idx >= N) {
        return;
    }

    // the row of input that this group of threads is responsible for
    const float *x = inp + idx * C;

    // mean
    float sum = 0.0f;
    for (int i = warp.thread_rank(); i < C; i += warp.size()) {
        sum += x[i];
    }
    sum = cg::reduce(warp, sum, cg::plus < float > {});
    float m = sum / C;
    if (warp.thread_rank() == 0 && mean != nullptr) {
        __stcs(mean + idx, m);
    }

    // rstd
    sum = 0.0f;
    for (int i = warp.thread_rank(); i < C; i += warp.size()) {
        float diff = x[i] - m;
        sum += diff * diff;
    }
    sum = cg::reduce(warp, sum, cg::plus < float > {});
    float s = rsqrtf(sum / C + 1e-5f);
    if (warp.thread_rank() == 0 && rstd != nullptr) {
        __stcs(rstd + idx, s);
    }

    // final normalization and scaling by weight/bias
    float *o = out + idx * C;
    for (int c = warp.thread_rank(); c < C; c += warp.size()) {
        // load and store using the .cs "streaming" hint to the compiler,
        // indicating that this data will not be reused soon, and can be streamed through the caches
        // this allows the threads to get more cache-hits for the (shared) weight and bias parameters
        float n = s * (__ldcs(x + c) - m);
        __stcs(o + c, n * weight[c] + bias[c]);
    }
}

// ----------------------------------------------------------------------------
// kernel launcher

void layernorm_forward1(float *out, float *mean, float *rstd,
                        const float *inp, const float *weight, const float *bias,
                        int B, int T, int C,
                        const int block_size) {
    const int N = B * T;
    const int grid_size = ceil_div(N, block_size);
    layernorm_forward_kernel1<<<grid_size, block_size>>>(out, mean, rstd, inp, weight, bias, N, C);
    cudaCheck(hipGetLastError());
}

void layernorm_forward2(float *out, float *mean, float *rstd,
                        const float *inp, const float *weight, const float *bias,
                        int B, int T, int C,
                        const int block_size) {
    int N = B * T;
    // in mean and rstd, threads cooperate within blocks via reductions
    mean_kernel<<<B * T, block_size, block_size * sizeof(float)>>>(mean, inp, N, C, block_size);
    cudaCheck(hipGetLastError());
    rstd_kernel<<<B * T, block_size, block_size * sizeof(float)>>>(rstd, inp, mean, N, C, block_size);
    cudaCheck(hipGetLastError());
    // in the normalization, everything just gets flattened out
    const int block_size2 = 256;
    const int grid_size = ceil_div(B * T * C, block_size2);
    normalization_kernel<<<grid_size, block_size2>>>(out, inp, mean, rstd, weight, bias, B, T, C);
    cudaCheck(hipGetLastError());
}

void layernorm_forward3(float *out, float *mean, float *rstd,
                        const float *inp, const float *weight, const float *bias,
                        int B, int T, int C,
                        const int block_size) {
    assert(block_size % 32 == 0);
    const int N = B * T;
    const int grid_size = ceil_div(N * 32, block_size);
    layernorm_forward_kernel3<<<grid_size, block_size>>>(out, mean, rstd, inp, weight, bias, N, C);
    cudaCheck(hipGetLastError());
}

// kernel version dispatch
void layernorm_forward(int kernel_num,
                       float *out, float *mean, float *rstd,
                       const float *inp, const float *weight, const float *bias,
                       int B, int T, int C,
                       const int block_size) {
    switch (kernel_num) {
        case 1:
            layernorm_forward1(out, mean, rstd, inp, weight, bias, B, T, C, block_size);
            break;
        case 2:
            layernorm_forward2(out, mean, rstd, inp, weight, bias, B, T, C, block_size);
            break;
        case 3:
            layernorm_forward3(out, mean, rstd, inp, weight, bias, B, T, C, block_size);
            break;
        default:
            printf("Invalid kernel number\n");
            exit(1);
    }
}

// ----------------------------------------------------------------------------

int main(int argc, char **argv) {
    srand(0);

    int B = 8;
    int T = 1024;
    int C = 768;

    int deviceIdx = 0;
    cudaCheck(hipSetDevice(deviceIdx));

    // create host memory of random numbers
    float *out = (float *) malloc(B * T * C * sizeof(float));
    float *mean = (float *) malloc(B * T * sizeof(float));
    float *rstd = (float *) malloc(B * T * sizeof(float));
    float *inp = make_random_float(B * T * C);
    float *weight = make_random_float(C);
    float *bias = make_random_float(C);

    // move to GPU
    float *d_out;
    float *d_mean;
    float *d_rstd;
    float *d_inp;
    float *d_weight;
    float *d_bias;
    cudaCheck(hipMalloc(&d_out, B * T * C * sizeof(float)));
    cudaCheck(hipMalloc(&d_mean, B * T * sizeof(float)));
    cudaCheck(hipMalloc(&d_rstd, B * T * sizeof(float)));
    cudaCheck(hipMalloc(&d_inp, B * T * C * sizeof(float)));
    cudaCheck(hipMalloc(&d_weight, C * sizeof(float)));
    cudaCheck(hipMalloc(&d_bias, C * sizeof(float)));
    cudaCheck(hipMemcpy(d_inp, inp, B * T * C * sizeof(float), hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(d_weight, weight, C * sizeof(float), hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(d_bias, bias, C * sizeof(float), hipMemcpyHostToDevice));

    // read kernel_num from command line
    int kernel_num = 2;
    if (argc > 1) {
        kernel_num = atoi(argv[1]);
    }
    printf("Using kernel %d\n", kernel_num);

    int block_sizes[] = {32, 64, 128, 256, 512, 1024};
    float *out_gpu = (float *) malloc(B * T * C * sizeof(float));
    float *mean_gpu = (float *) malloc(B * T * sizeof(float));
    float *rstd_gpu = (float *) malloc(B * T * sizeof(float));

    layernorm_forward_cpu(out, mean, rstd, inp, weight, bias, B, T, C);

    // check the correctness of the kernel at all block sizes
    for (int j = 0; j < sizeof(block_sizes) / sizeof(int); j++) {
        int block_size = block_sizes[j];
        printf("Checking block size %d.\n", block_size);

        layernorm_forward(kernel_num, d_out, d_mean, d_rstd, d_inp, d_weight, d_bias, B, T, C, block_size);

        validate_result(d_out, out, "out", B * T * C, 1e-5f);
        validate_result(d_mean, mean, "mean", B * T, 1e-5f);
        validate_result(d_rstd, rstd, "rstd", B * T, 1e-5f);
    }

    printf("All results match. Starting benchmarks.\n\n");

    // time the kernel at different block sizes
    for (int j = 0; j < sizeof(block_sizes) / sizeof(int); j++) {
        int block_size = block_sizes[j];

        int repeat_times = 1000;
        float elapsed_time = benchmark_kernel(repeat_times, layernorm_forward,
                                              kernel_num, d_out, d_mean, d_rstd, d_inp, d_weight, d_bias,
                                              B, T, C, block_size);

        // napkin math: estimate the memory bandwidth achieved
        // e.g. A100 40GB PCIe is advertised at 1,555GB/s
        long memory_ops = (2 * B * T * C) * 4; // *4 for float
        float memory_bandwidth = memory_ops / elapsed_time / 1e6;

        printf("block_size %4d | time %.4f ms | bandwidth %.2f GB/s\n", block_size, elapsed_time, memory_bandwidth);
    }

    // free memory
    free(out);
    free(mean);
    free(rstd);
    free(inp);
    free(weight);
    free(bias);
    cudaCheck(hipFree(d_out));
    cudaCheck(hipFree(d_mean));
    cudaCheck(hipFree(d_rstd));
    cudaCheck(hipFree(d_inp));
    cudaCheck(hipFree(d_weight));
    cudaCheck(hipFree(d_bias));

    return 0;
}